#include "hip/hip_runtime.h"
//���̺� �ڵ� ����

#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <vector>
#include "hip/hip_runtime.h"
#include ""

#define TILE_WIDTH 8
#define SIZE 512

void MatrixMultiplication(const float* M, const float* N, float* P, int Width);
__global__ void MatrixMulKernel(float* Md, float* Nd, float* Pd, int Width);

class Timer {
	struct timespec s_;
public:
	clock_t start, end;
	Timer() { tic(); }
	void tic() {
		start = clock();
	}

	double toc() {
		end = clock();
		return (double)(end - start);
	}
};

// Please optimize this function
void matmult(int m, int n, int k, const float* mat_a, const float* mat_b, float* mat_c)
{
	/*
		== input ==
		mat_a: m x k matrix
		mat_b: k x n matrix
		== output ==
		mat_c: m x n matrix (output)
	*/

	MatrixMultiplication(mat_a, mat_b, mat_c, SIZE);
}

void genmat(int n, int m, std::vector<float>& mat)
{
	srand(time(0));
	mat.resize(n * m);
	for (int i = 0; i < mat.size(); i++) mat[i] = rand() % 100;
}

void dumpmat(int n, int m, std::vector<float>& mat)
{
	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < m; j++)
			printf("%f ", mat[i * m + j]);
		printf("\n");
	}
}

int main(int argc, char** argv)
{
	std::vector<float> mat_a;
	std::vector<float> mat_b;
	std::vector<float> mat_c;

	genmat(SIZE, SIZE, mat_a);
	genmat(SIZE, SIZE, mat_b);
	genmat(SIZE, SIZE, mat_c);

	Timer t;
	double elapsed = 0;
	const int iteration = 10000;

	for (int i = 0; i < iteration; i++)
	{
		t.tic();
		matmult(SIZE, SIZE, SIZE, &mat_a[0], &mat_b[0], &mat_c[0]);
		elapsed += t.toc();
	}
	dumpmat(SIZE, SIZE, mat_a);
	dumpmat(SIZE, SIZE, mat_c);
	elapsed /= (CLOCKS_PER_SEC);
	printf("%lf s\n", elapsed);
	return 0;
}

void MatrixMultiplication(const float* M, const float* N, float* P, int Width) {
	int size = Width * Width * sizeof(int);
	float *Md, *Nd, *Pd;


	// Transfer M and N to device memory
	hipMalloc((void**)&Md, size);
	hipMemcpy(Md, M, size, hipMemcpyHostToDevice);
	hipMalloc((void**)&Nd, size);
	hipMemcpy(Nd, N, size, hipMemcpyHostToDevice);

	// Allocate P on the device
	hipMalloc((void**)&Pd, size);

	// Setup the execution configuration
	dim3 dimGrid(Width, Width);
	dim3 dimBlock(Width, Width);

	// Launch the device computation threads!
	MatrixMulKernel << <dimGrid, dimBlock >> > (Md, Nd, Pd, Width);

	// Transfer P from device to host
	hipMemcpy(P, Pd, size, hipMemcpyDeviceToHost);
	// Free device matrices
	hipFree(Md);
	hipFree(Nd);
	hipFree(Pd);

}


__global__ void MatrixMulKernel(float* Md, float* Nd, float* Pd, int Width) {
	// Calculate the row index of the Pd element and M
	int Row = blockIdx.y * blockDim.y + threadIdx.y;
	// Calculate the column index of Pd and N
	int Col = blockIdx.x * blockDim.x + threadIdx.x;

	int Pvalue = 0;
	// each thread computes one element of the bolck sub-matrix
	for (int k = 0; k < Width; ++k)
		Pvalue += Md[Row * Width + k] * Nd[k * Width + Col];

	Pd[Row * Width + Col] = Pvalue;
}